#include "hip/hip_runtime.h"
#include <cassert>
#include "breaker.cuh"
#include <stdio.h>

__device__ int strlen_d(const char *str) {
    int len = 0;
    while (*str) {
        len++;
        str++;
    }

    return len;
}

__device__ bool strcmp_d(const char *str1, const char *str2) {
    while (*str1 && *str2) {
        if (*str1 != *str2) return false;
        str1++;
        str2++;
    }

    return bool(*str1 == *str2);
}

__device__ bool arrcmp_d(const uint8_t *arr1, const uint8_t *arr2, const int len) {
    for(int i = 0 ; i < len ; i ++, arr1++, arr2++){
        if (*arr1 != *arr2) return false;
    }

    return true;
}

__global__ void breaker_kernel(const char *dict, const uint8_t *hashed, const int max_len,
                               hash_func hash, const int hash_len) {
    int tid_x = threadIdx.x;
    int tid_y = threadIdx.y;
    int gid_x = blockIdx.x;
    int gid_y = blockIdx.y;
    int tid = tid_y * blockDim.y + tid_x;
    int gid = gid_y * gridDim.y + gid_x;
    int threads_per_block = blockDim.x * blockDim.y;
    int idx = threads_per_block * gid + tid;
    int total_num_threads = gridDim.x * gridDim.y * threads_per_block;

    int dict_len = strlen_d(dict) + 1;  // including '/0'

    // int goal_len = 5;
    for (int goal_len = 2; goal_len <= max_len ; goal_len ++){
        int N = pow(dict_len - 1, goal_len - 1);
        int round = (N + total_num_threads - 1) / total_num_threads;
        char *orig_pwd = new char[goal_len];
        // printf("%s %d\n", dict, strlen_d(dict));
        for (int r = 0; r < round; r++) {
            // printf("stat %d\n", tid);
            int num = idx + r * total_num_threads;
            if (num > N) break;
            int base = 1;
            memset(orig_pwd, 0, goal_len);
            // gen
            for (int j = 0; j < goal_len - 1; j++) {
                orig_pwd[j] = dict[num / base % (dict_len - 1)];
                base *= dict_len - 1;
            }
            
            // hashing
            uint8_t hashed_pwd_int[16];
            md5((uint8_t *)orig_pwd, goal_len - 1, hashed_pwd_int);

            // printf("The orig_pwd is [%s].\n", orig_pwd);

            if (arrcmp_d(hashed, hashed_pwd_int,hash_len )) {
                printf("Password Hacked: %s\n", orig_pwd);
                delete[] orig_pwd;
                asm("trap;");
                return;
            }
            // if (strcmp_d(orig_pwd, hashed_str)) {
            //     printf("\nThe cracked password is [%s].\n", orig_pwd);
            //     // asm("trap;");
            //     return;
            // }
        }
        delete[] orig_pwd;
    }
}

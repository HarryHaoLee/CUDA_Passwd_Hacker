#include "hip/hip_runtime.h"
#include <cassert>
#include "breaker.cuh"

__device__ int strlen_d(char *str) {
    int len = 0;
    char *p = str;
    while (*p) {
        len++;
        p++;
    }

    return len;
}

__device__ bool strcmp_d(char *str1, char *str2) {
    char *p1 = str1;
    char *p2 = str2;
    
    while (*p1 && *p2) {
        if (*p1 != *p2) return false;
        p1++;
        p2++;
    }

    return bool(*p1 == *p2);
}

__global__ void breaker_kernel(char *dict, char *goal, int goal_len,
                               hash_func hash, int hash_len) {
    int tid_x = threadIdx.x;
    int tid_y = threadIdx.y;
    int gid_x = blockIdx.x;
    int gid_y = blockIdx.y;
    int tid = tid_y * blockDim.y + tid_x;
    int gid = gid_y * gridDim.y + gid_x;
    int threads_per_block = blockDim.x * blockDim.y;
    int idx = threads_per_block * gid + tid;
    int total_num_threads = gridDim.x * gridDim.y * threads_per_block;

    int dict_len = strlen_d(dict) + 1;  // including '/0'
    int N = pow(dict_len - 1, goal_len - 1);
    int round = (N + total_num_threads - 1) / total_num_threads;
    // printf("%s %d\n", dict, strlen_d(dict));
    for (int r = 0; r < round; r++) {
        // printf("stat %d\n", tid);
        int num = idx + r * total_num_threads;
        if (num > N) return;
        int base = 1;
        char *orig_pwd = new char[goal_len];
        memset(orig_pwd, 0, goal_len);
        // gen
        for (int j = 0; j < goal_len - 1; j++) {
            orig_pwd[j] = dict[num / base % (dict_len - 1)];
            base *= dict_len - 1;
        }
        // hashing
        // char *hashed_pwd = new char[hash_len];
        // memset(hashed_pwd, 0, hash_len);
        // hash(orig_pwd, goal_len, hashed_pwd);
        // cmp
        // if (strcmp_d(hashed_pwd, goal)) {
        //     printf("Password Hacked: %s", orig_pwd);
        // assert(0);
        // }
        if (strcmp_d(orig_pwd, goal)) {
            printf("\nThe cracked password is [%s].\n", orig_pwd);
            // asm("trap;");
            uint8_t result[16];
            md5((uint8_t *)orig_pwd, goal_len - 1, result);
            for (int i = 0; i < 16; i++) {
                printf("%2.2x", result[i]);
            }
            printf("\n");
            return;
        }
        delete[] orig_pwd;
        // delete [] hashed_pwd;
    }
}

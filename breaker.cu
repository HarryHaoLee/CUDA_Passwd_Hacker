#include "hip/hip_runtime.h"
#include <iostream>
#include <cstring>
#include "breaker.cuh"
#include "hash.h"

using std::endl;
using std::cout;

__global__ void breaker_kernel(char* dict, char* goal, int goal_len, hash_func hash, int hash_len) {
    int tid_x = threadIdx.x;
    int tid_y = threadIdx.y;
    int gid_x = blockIdx.x;
    int gid_y = blockIdx.y;
    int tid = tid_x * blockDim.x + tid_y;
    int gid = gid_x * gridDim.x + gid_y;
    int idx = threads_per_block * gid + tid;
    int threads_per_block = blockDim.x * blockDim.y;
    int total_num_threads = gridDim.x * gridDim.y + threads_per_block;
    
    int dict_len = strlen(dict) + 1; // including '/0'
    int N = pow(goal_len - 1, dict_len - 1);
    int round = (N + total_num_threads - 1) / total_num_threads;
    
    for (int r = 0; r < round; r++) {
        num = tid + r * total_num_threads;
        if (num > N) return;
        int base = 1;
        char orig_pwd[goal_len] = {0};
        // gen
        for (int j = 0; j < goal_len - 1; j++) {
            orig_pwd[j] = dict[num / base % (dict_len - 1)];
            base *= dict_len - 1;
        }
        // hashing
        char hashed_pwd[hash_len] = {0};
        hash(pwd, hashed_pwd);
        // cmp
        if (!strcmp(hashed_pwd, goal)) {
            cout << "Password Hacked: " << orig_pwd << endl;
            assert(0);
        }
    }
}


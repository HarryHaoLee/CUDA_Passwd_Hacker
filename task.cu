#include <iostream>
#include <cstdlib>
#include "breaker.cuh"

using std::cout;
using std::endl;

int main(int argc, char** argv) {
    unsigned int n = atoi(argv[1]);

    hipEvent_t start;
    hipEvent_t stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    
    hipEventRecord(start);
    // call the kernel
    kernel_placeholder()
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    
    // Get the elapsed time in milliseconds
    float ms;
    hipEventElapsedTime(&ms, start, stop);
    
    cout << ms << endl;

    return 0;
}

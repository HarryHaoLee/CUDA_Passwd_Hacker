#include <iostream>
#include <cstdlib>
#include <cstring>
#include <string.h>
#include "breaker.cuh"
#include "hash.cuh"

using std::cout;
using std::endl;

int main(int argc, char** argv) {
    unsigned int num_grids = atoi(argv[1]);
    unsigned int num_threads = atoi(argv[2]);
    char *tmp = argv[3];
    char _tmp[3] = {0};
    // print device info
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);
    
    cout << "Device Name: " << prop.name << endl;
    cout << "Max Threads Per Block: " << prop.maxThreadsPerBlock << endl;

    hipEvent_t start;
    hipEvent_t stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // setting up dictionary, goal & hash function
    char dict[] = "1234567890abcdefghijklmnopqrstuvwxyzABCDEFGHIJKLMNOPQRSTUVWXYZ";
    const int dict_len = strlen(dict) + 1;
    int max_len = 4 + 1;
    hash_func hash = identity_mapping;
    int hash_len = 16; // MD5

    char *dict_d;
    uint8_t *hashed_d;

    hipMallocManaged(&dict_d, dict_len * sizeof(char));
    hipMallocManaged(&hashed_d, 16 * sizeof(uint8_t));

    memcpy(dict_d, dict, dict_len * sizeof(char));
    cout<<"The input hash string is: "<<tmp<<endl;
    for (int i = 0 ; i < 16 ; i ++){
        strncpy(_tmp, tmp+i*2, 2);
        sscanf(_tmp, "%x", &hashed_d[i]);
    }

    hipEventRecord(start);
    // call the kernel
    breaker_kernel<<<num_grids, num_threads>>>(dict_d, hashed_d, max_len, hash, hash_len);
    hipError_t cudaerr = hipDeviceSynchronize();
    if (cudaerr != hipSuccess)
        printf(">>> kernel launch failed with error \"%s\".\n",
            hipGetErrorString(cudaerr));
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    
    // Get the elapsed time in milliseconds
    float ms;
    hipEventElapsedTime(&ms, start, stop);
    
    cout << "Password cracked in ["<< ms <<"] ms." << endl;

    return 0;
}

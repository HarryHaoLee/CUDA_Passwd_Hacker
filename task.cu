#include <iostream>
#include <cstdlib>
#include <cstring>
#include "breaker.cuh"
#include "hash.cuh"

using std::cout;
using std::endl;

int main(int argc, char** argv) {
    unsigned int num_grids = atoi(argv[1]);
    unsigned int num_threads = atoi(argv[2]);
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);
    
    cout << "Device Name: " << prop.name << endl;
    cout << "Max Threads Per Block: " << prop.maxThreadsPerBlock << endl;

    hipEvent_t start;
    hipEvent_t stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // setting up dictionary, goal & hash function
    char dict[] = "1234567890";
    char goal[] = "759";
    int dict_len = strlen(dict) + 1;
    int goal_len = strlen(goal) + 1;
    hash_func hash = identity_mapping;
    int hash_len = goal_len + 1;

    char *dict_d, *goal_d;
    hipMallocManaged(&dict_d, dict_len * sizeof(char));
    hipMallocManaged(&goal_d, goal_len * sizeof(char));

    for (int i = 0; i < dict_len; i++)
        dict_d[i] = dict[i];
    for (int i = 0; i < goal_len; i++)
        goal_d[i] = goal[i];
    
    hipEventRecord(start);
    // call the kernel
    breaker_kernel<<<num_grids, num_threads>>>(dict_d, goal_d, goal_len, hash, hash_len);
    hipError_t cudaerr = hipDeviceSynchronize();
    if (cudaerr != hipSuccess)
        printf(">>> kernel launch failed with error \"%s\".\n",
            hipGetErrorString(cudaerr));
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    
    // Get the elapsed time in milliseconds
    float ms;
    hipEventElapsedTime(&ms, start, stop);
    
    cout << "Password cracked in ["<< ms <<"] ms." << endl;

    return 0;
}

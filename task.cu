#include <iostream>
#include <cstdlib>
#include <cstring>
#include "breaker.cuh"
#include "hash.h"

using std::cout;
using std::endl;

int main(int argc, char** argv) {
    unsigned int num_grids = atoi(argv[1]);
    unsigned int num_threads = atoi(argv[2]);
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);
    
    cout << "Device Name: " << prop.name << endl;
    cout << "Max Threads Per Block: " << prop.maxThreadsPerBlock << endl;

    hipEvent_t start;
    hipEvent_t stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // setting up dictionary, goal & hash function
    char *dict = "1234567890";
    char *goal = "759";
    int goal_len = strlen(goal) + 1;
    hash_func hash = identity_mapping;
    int hash_len = goal_len + 1;
    
    hipEventRecord(start);
    // call the kernel
    breaker_kernel<<<num_grids, num_threads>>>(dict, goal, goal_len, hash, hash_len);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    
    // Get the elapsed time in milliseconds
    float ms;
    hipEventElapsedTime(&ms, start, stop);
    
    cout << ms << endl;

    return 0;
}


#include <hip/hip_runtime.h>
#include <stdint.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

__device__ uint32_t k[64] = {0};

// K specifies the per-round shift amounts
__device__ const uint32_t K[] = {
    7, 12, 17, 22, 7, 12, 17, 22, 7, 12, 17, 22, 7, 12, 17, 22,
    5, 9,  14, 20, 5, 9,  14, 20, 5, 9,  14, 20, 5, 9,  14, 20,
    4, 11, 16, 23, 4, 11, 16, 23, 4, 11, 16, 23, 4, 11, 16, 23,
    6, 10, 15, 21, 6, 10, 15, 21, 6, 10, 15, 21, 6, 10, 15, 21};

// leftrotate function definition
__device__ uint32_t leftrotate(uint32_t x, uint32_t C) {
    return (((x) << (C)) | ((x) >> (32 - (C))));
}

__device__ void append_bytes(uint32_t val, uint8_t *outputs) {
    outputs[0] = (uint8_t)val;
    outputs[1] = (uint8_t)(val >> 8);
    outputs[2] = (uint8_t)(val >> 16);
    outputs[3] = (uint8_t)(val >> 24);
}

__device__ uint32_t append_int(const uint8_t *inputs) {
    return (uint32_t)inputs[0] | ((uint32_t)inputs[1] << 8) |
           ((uint32_t)inputs[2] << 16) | ((uint32_t)inputs[3] << 24);
}

__device__ void md5(const uint8_t *orig_msg, size_t orig_len,
                    uint8_t *digest) {
    // Use binary integer part of the sines of integers (Radians) as constants:
    for (int i = 0; i < 64; i++) {
        k[i] = (uint32_t)(abs(sin(i + 1.0)) * ((long long)1 << 32));
    }

    // Initialize variables:
    uint32_t a0 = 0x67452301;
    uint32_t b0 = 0xefcdab89;
    uint32_t c0 = 0x98badcfe;
    uint32_t d0 = 0x10325476;

    size_t new_len, offset;
    uint32_t M[16];
    uint32_t A, B, C, D, F, g;

    // append "0" bit until message length in bits ≡ 448 (mod 512)
    for (new_len = orig_len + 1; new_len % (512 / 8) != 448 / 8; new_len++);
    uint8_t *message = (uint8_t *)malloc(new_len + 8);
    memcpy(message, orig_msg, orig_len);
    // Pre-processing: adding a single 1 bit
    message[orig_len] = 0x80;
    // Pre-processing: padding with zeros
    for (offset = orig_len + 1; offset < new_len; offset++){
        message[offset] = 0;
    }

    // append length mod (2^64) to message
    append_bytes(orig_len * 8, message + new_len);
    // address the overflow part
    append_bytes(orig_len >> 29, message + new_len + 4);

    // Process the message in successive 512-bit chunks:
    // for each 512-bit chunk of message:
    for (offset = 0; offset < new_len; offset += (512 / 8)) {
        // break chunk into sixteen 32-bit words w[j], 0 ≤ j ≤ 15
        for (int i = 0; i < 16; i++) {
            M[i] = append_int(message + offset + i * 4);
        }

        // Initialize hash value for this chunk:
        A = a0;
        B = b0;
        C = c0;
        D = d0;

        // Main loop:
        for (int i = 0; i < 64; i++) {
            if (i < 16) {
                F = (B & C) | ((~B) & D);
                g = i;
            } else if (i < 32) {
                F = (D & B) | ((~D) & C);
                g = (5 * i + 1) % 16;
            } else if (i < 48) {
                F = B ^ C ^ D;
                g = (3 * i + 5) % 16;
            } else {
                F = C ^ (B | (~D));
                g = (7 * i) % 16;
            }
            // Be wary of the below definitions of a,b,c,d
            F = A + F + k[i] + M[g];  // M[g] must be a 32-bits block
            A = D;
            D = C;
            C = B;
            B = B + leftrotate(F, K[i]);
        }

        // Add this chunk's hash to result so far:
        a0 += A;
        b0 += B;
        c0 += C;
        d0 += D;
    }

    // cleanup
    free(message);

    // var char digest[16] := a0 append b0 append c0 append d0 //(Output is in
    // little-endian)
    append_bytes(a0, digest);
    append_bytes(b0, digest + 4);
    append_bytes(c0, digest + 8);
    append_bytes(d0, digest + 12);
}

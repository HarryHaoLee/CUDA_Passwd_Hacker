
#include <hip/hip_runtime.h>
#include <stdint.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

// Constants are the integer part of the sines of integers (in radians) * 2^32.
__device__ uint32_t k[64] = {0};

// r specifies the per-round shift amounts
__device__ const uint32_t r[] = {
    7, 12, 17, 22, 7, 12, 17, 22, 7, 12, 17, 22, 7, 12, 17, 22,
    5, 9,  14, 20, 5, 9,  14, 20, 5, 9,  14, 20, 5, 9,  14, 20,
    4, 11, 16, 23, 4, 11, 16, 23, 4, 11, 16, 23, 4, 11, 16, 23,
    6, 10, 15, 21, 6, 10, 15, 21, 6, 10, 15, 21, 6, 10, 15, 21};

// leftrotate function definition
__device__ uint32_t leftrotate(uint32_t x, uint32_t C) {
    return (((x) << (C)) | ((x) >> (32 - (C))));
}

__device__ void to_bytes(uint32_t val, uint8_t *bytes) {
    bytes[0] = (uint8_t)val;
    bytes[1] = (uint8_t)(val >> 8);
    bytes[2] = (uint8_t)(val >> 16);
    bytes[3] = (uint8_t)(val >> 24);
}

__device__ uint32_t to_int32(const uint8_t *bytes) {
    return (uint32_t)bytes[0] | ((uint32_t)bytes[1] << 8) |
           ((uint32_t)bytes[2] << 16) | ((uint32_t)bytes[3] << 24);
}

__device__ void md5(const uint8_t *initial_msg, size_t initial_len,
                    uint8_t *digest) {
    // Use binary integer part of the sines of integers (Radians) as constants:
    for (int i = 0; i < 64; i++) {
        k[i] = (uint32_t)(abs(sin(i + 1.0)) * ((long long)1 << 32));
    }

    // Initialize variables:
    uint32_t a0 = 0x67452301;
    uint32_t b0 = 0xefcdab89;
    uint32_t c0 = 0x98badcfe;
    uint32_t d0 = 0x10325476;

    uint8_t *message = NULL;

    size_t new_len, offset;
    uint32_t M[16];
    uint32_t A, B, C, D, F, g;

    // Pre-processing:
    // append "1" bit to message
    // append "0" bits until message length in bits ≡ 448 (mod 512)
    // append length mod (2^64) to message
    for (new_len = initial_len + 1; new_len % (512 / 8) != 448 / 8; new_len++);

    message = (uint8_t *)malloc(new_len + 8);
    memcpy(message, initial_msg, initial_len);
    message[initial_len] = 0x80;  // append the "1" bit
    for (offset = initial_len + 1; offset < new_len; offset++){
        message[offset] = 0;  // append "0" bits
    }

    // append the len in bits at the end of the buffer.
    to_bytes(initial_len * 8, message + new_len);
    // address the overflow part
    to_bytes(initial_len >> 29, message + new_len + 4);

    // Process the message in successive 512-bit chunks:
    // for each 512-bit chunk of message:
    for (offset = 0; offset < new_len; offset += (512 / 8)) {
        // break chunk into sixteen 32-bit words w[j], 0 ≤ j ≤ 15
        for (int i = 0; i < 16; i++) {
            M[i] = to_int32(message + offset + i * 4);
        }

        // Initialize hash value for this chunk:
        A = a0;
        B = b0;
        C = c0;
        D = d0;

        // Main loop:
        for (int i = 0; i < 64; i++) {
            if (i < 16) {
                F = (B & C) | ((~B) & D);
                g = i;
            } else if (i < 32) {
                F = (D & B) | ((~D) & C);
                g = (5 * i + 1) % 16;
            } else if (i < 48) {
                F = B ^ C ^ D;
                g = (3 * i + 5) % 16;
            } else {
                F = C ^ (B | (~D));
                g = (7 * i) % 16;
            }

            F = A + F + k[i] + M[g];  // M[g] must be a 32-bits block
            A = D;
            D = C;
            C = B;
            B = B + leftrotate(F, r[i]);
        }

        // Add this chunk's hash to result so far:
        a0 += A;
        b0 += B;
        c0 += C;
        d0 += D;
    }

    // cleanup
    free(message);

    // var char digest[16] := a0 append b0 append c0 append d0 //(Output is in
    // little-endian)
    to_bytes(a0, digest);
    to_bytes(b0, digest + 4);
    to_bytes(c0, digest + 8);
    to_bytes(d0, digest + 12);
}
